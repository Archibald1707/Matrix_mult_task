#include "hip/hip_runtime.h"
#ifndef CUDA_MUL_H
#define CUDA_MUL_H

#include "Matrix.h"
#include "FlatMatrix.h"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void matrixMulKernel(const double* A, const double* B, double* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        double sum = 0.0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

Matrix multiply_cuda(const Matrix& A, const Matrix& B) {
    int N = A.rows;
    size_t size = N * N * sizeof(double);

    double* h_A = new double[N * N];
    double* h_B = new double[N * N];
    double* h_C = new double[N * N];

    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j) {
            h_A[i * N + j] = A[i][j];
            h_B[i * N + j] = B[i][j];
        }

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);
    matrixMulKernel<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    Matrix C(N, N);
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            C[i][j] = h_C[i * N + j];

    delete[] h_A;
    delete[] h_B;
    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}

__global__ void matrixMulKernelFlat(const double* A, const double* B, double* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        double sum = 0.0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[k * N + col];
        }
        C[row * N + col] = sum;
    }
}

FlatMatrix multiply_flat_cuda(const FlatMatrix& A, const FlatMatrix& B) {
    int N = A.rows;
    size_t size = N * N * sizeof(double);

    const double* h_A = A.data.data();
    const double* h_B = B.data.data();

    double* d_A;
    double* d_B;
    double* d_C;

    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);

    matrixMulKernelFlat<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    double* h_C = new double[N * N];
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    FlatMatrix C(N, N);
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            C(i, j) = h_C[i * N + j];

    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}

__global__ void matrixMulKernelTransposed(const double* A, const double* B, double* C, int N) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < N && col < N) {
        double sum = 0.0;
        for (int k = 0; k < N; ++k) {
            sum += A[row * N + k] * B[col * N + k];
        }
        C[row * N + col] = sum;
    }
}

FlatMatrix multiply_flat_transposed_cuda(FlatMatrix& A, FlatMatrix& B) {
    int N = A.rows;
    size_t size = N * N * sizeof(double);

    const double* h_A = A.data.data();
    const double* h_B = B.data.data();
    double* h_C = new double[N * N];

    double *d_A, *d_B, *d_C;
    hipMalloc(&d_A, size);
    hipMalloc(&d_B, size);
    hipMalloc(&d_C, size);

    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    dim3 threadsPerBlock(16, 16);
    dim3 numBlocks((N + 15) / 16, (N + 15) / 16);
    matrixMulKernelTransposed<<<numBlocks, threadsPerBlock>>>(d_A, d_B, d_C, N);

    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);

    FlatMatrix C(N, N);
    for (int i = 0; i < N; ++i)
        for (int j = 0; j < N; ++j)
            C(i, j) = h_C[i * N + j];

    delete[] h_C;
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    return C;
}

#endif
